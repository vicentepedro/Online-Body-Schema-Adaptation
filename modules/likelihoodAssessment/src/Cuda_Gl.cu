#include "hip/hip_runtime.h"
/*
 * Copyright: (C) 2017 VisLab, Institute for Systems and Robotics,
 *                Instituto Superior Técnico, Universidade de Lisboa, Lisbon, Portugal
 * Author: Pedro Vicente <pvicente@isr.tecnico.ulisboa.pt>
 * CopyPolicy: Released under the terms of the GNU GPL v3.0.
 *
 */

#include <hip/hip_runtime_api.h>


texture<float4, 2, hipReadModeElementType> inTex;

__global__ void CuDeviceArrayCopyFromTexture( float3* dst, int dstStep, int width, int height )     
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ( x > width || y > height ) return;

    float4 res = tex2D(inTex, x, y);
    float3* row_y = (float3*)((char*)dst + y * dstStep);
    row_y[x] = make_float3(res.x, res.y, res.z);
}
// round up n/m
inline int iDivUp(int n, int m)
{
    return (n + m - 1) / m;
}

extern "C" void DeviceArrayCopyFromTexture( float3* dst, int dstStep, int width, int height ) 
{
    dim3 threads( 64, 1 );
    dim3 grid = dim3( iDivUp( width, threads.x ), height/threads.y );
    CuDeviceArrayCopyFromTexture <<< grid, threads >>> ( dst, dstStep, width, height );
}

extern "C" void BindToTexture( hipArray *cuArr )
{
     checkCudaErrors( hipBindTextureToArray( inTex, cuArr ) );
}
